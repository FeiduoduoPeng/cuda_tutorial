#include "hip/hip_runtime.h"
#include "vector_add.h"

__global__ void add(int *a, int *b, int *c){
    int tid = blockIdx.x;
    if(tid<N){
        c[tid] = a[tid] +b[tid];
    }
}

void kernel_wrapper(int *a, int *b, int *c){
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void**) &dev_a, N*sizeof(int));
    hipMalloc((void**) &dev_b, N*sizeof(int));
    hipMalloc((void**) &dev_c, N*sizeof(int));

    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);

    add<<<N,1>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);    
    hipFree(dev_b);    
    hipFree(dev_c);   
}